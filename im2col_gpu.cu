#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"

#include <cstdio>

__device__ int calcsum(int o, dim3 i, dim3 d)
{
	return (((o * d.z) + i.z) * d.y + i.y) * d.x + i.x;
}

__global__ void im2col(const float *const A, float *const B, int const m, int const n)
{
	int offset = calcsum(calcsum(0, blockIdx, gridDim), threadIdx, blockDim);
	int pffset = ((blockIdx.z * blockDim.z + threadIdx.z) * n + blockIdx.y + threadIdx.y) * m + blockIdx.x + threadIdx.x;

	B[offset] = A[pffset];
}

// TODO : Modify Error Messages, Add Stride Support
void mexFunction(int nlhs, mxArray *plhs[],
	int nrhs, mxArray const *prhs[])
{
	mxGPUArray const *A;
	mxGPUArray *B;
	float const *d_A;
	float *d_B;
	char const * const errId = "parallel:gpu:im2col_gpu:InvalidInput";
	char const * const errMsg = "Invalid input to MEX file.";
	mwSize nda;

	mxInitGPU();

	if ((nrhs < 2) || !(mxIsGPUArray(prhs[0]))) {
		mexErrMsgIdAndTxt(errId, errMsg);
	}

	A = mxGPUCreateFromMxArray(prhs[0]);
	nda = mxGPUGetNumberOfDimensions(A);

	if (nda < 2 || nda > 4) {
		mexErrMsgIdAndTxt(errId, errMsg);
	} 
	
	if (mxGPUGetClassID(A) != mxSINGLE_CLASS) {
		mexErrMsgIdAndTxt(errId, errMsg);
	}

	if (mxGetNumberOfElements(prhs[1]) < 1) {
		mexErrMsgIdAndTxt(errId, errMsg);
	}

	int m, n;

	if (mxGetNumberOfElements(prhs[1]) == 1) {
		m = n = mxGetScalar(prhs[1]);
	} else {
		auto *p = mxGetPr(prhs[1]);

		m = p[0];
		n = p[1];
	}

	auto *asz = mxGPUGetDimensions(A);

	mwSize mm = asz[0];
	mwSize nn = asz[1];
	mwSize c;
	mwSize N;

	if (nda < 3) {
		c = 1;
	} else {
		c = asz[2];
	}

	if (nda < 4) {
		N = 1;
	} else {
		N = asz[3];
	}

	mwSize s1 = mm - m + 1;
	mwSize s2 = nn - n + 1;

	mwSize bsz[6] = {m, n, c, s1, s2, N};

	d_A = (float const *)(mxGPUGetDataReadOnly(A));

	B = mxGPUCreateGPUArray(6, bsz,
		mxGPUGetClassID(A),
		mxGPUGetComplexity(A),
		MX_GPU_DO_NOT_INITIALIZE);
	d_B = (float *)(mxGPUGetData(B));

	im2col<<<dim3(bsz[3], bsz[4], bsz[5]), dim3(bsz[0], bsz[1], bsz[2])>>>(d_A, d_B, mm, nn);

	plhs[0] = mxGPUCreateMxArrayOnGPU(B);

	mxGPUDestroyGPUArray(A);
	mxGPUDestroyGPUArray(B);
}

